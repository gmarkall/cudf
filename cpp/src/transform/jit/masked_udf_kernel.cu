#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Include Jitify's cstddef header first
#include <cstddef>

#include <cuda/std/climits>
#include <cuda/std/cstddef>
#include <cuda/std/limits>
#include <cuda/std/type_traits>

#include <cudf/wrappers/durations.hpp>
#include <cudf/wrappers/timestamps.hpp>

#include <transform/jit/operation-udf.hpp>

#include <cudf/types.hpp>
#include <cudf/wrappers/timestamps.hpp>
#include <cudf/utilities/bit.hpp>

#include <tuple>
#include <cuda/std/tuple>

namespace cudf {
namespace transformation {
namespace jit {

template <typename T>
struct Masked {
  T value;
  bool valid;
};

template <typename TypeIn, typename MaskType, typename OffsetType>
__device__ auto make_args(cudf::size_type id,
                          TypeIn in_ptr,
                          MaskType in_mask,
                          OffsetType in_offset) 
{
    bool valid = in_mask ? cudf::bit_is_set(in_mask, in_offset + id) : true;
    return cuda::std::make_tuple(in_ptr[id], valid);
}

template <typename InType, typename MaskType, typename OffsetType, typename ... Arguments>
__device__ auto make_args(cudf::size_type id, 
                          InType in_ptr, 
                          MaskType in_mask,     // in practice, always cudf::bitmask_type const* 
                          OffsetType in_offset,  // in practice, always cudf::size_type
                          Arguments ... args) {

    bool valid = in_mask ? cudf::bit_is_set(in_mask, in_offset + id) : true;
    return cuda::std::tuple_cat(
        cuda::std::make_tuple(in_ptr[id], valid),
        make_args(id, args...)
    );
}

template <typename TypeOut, typename ... Arguments>
__global__
void generic_udf_kernel(cudf::size_type size, 
                        TypeOut* out_data, 
                        bool* out_mask, 
                        Arguments ... args)
{   

    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;
    int start = tid + blkid * blksz;
    int step = blksz * gridsz;

    Masked<TypeOut> output;
    for (cudf::size_type i=start; i<size; i+=step) {
      auto func_args = cuda::std::tuple_cat(
          cuda::std::make_tuple(&output.value),
          make_args(i, args...) // passed int64*, bool*, int64, int64*, bool*, int64
      );
      cuda::std::apply(GENERIC_OP, func_args);
      out_data[i] = output.value;
      out_mask[i] = output.valid;
    }

}


}  // namespace jit
}  // namespace transformation
}  // namespace cudf
